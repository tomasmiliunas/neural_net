#include "hip/hip_runtime.h"
#include "ann.h"



/*
__global__ void
kernel(int n, float *arr){

	volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;
	if(idx >= n) return;

        arr[idx] *= 2.0f;
}

void run_cuda_sample(){

  int deviceCount = 0;
  checkCudaErrors( hipGetDeviceCount(&deviceCount));
  if(deviceCount == 0){
    printf("*** there is no CUDE device\n");
    return;
  }

  checkCudaErrors( hipSetDevice(0) );

  int n = 11; // number of elements

  float *arr = new float[n];
  for(int i = 0; i < n; i++)
    arr[i] = i;

  int h = 4; // number of threads in block
  int g = (n + (h-n%h))/h; // number of grids

  printf("n=%d, h=%d, g=%d\n", n, h, g);





  int bc_arr = sizeof(float)*n;

  float *dv_arr = NULL;

  checkCudaErrors( hipMalloc((void **)&dv_arr, bc_arr) );

  checkCudaErrors( hipMemcpy(dv_arr, arr, bc_arr, hipMemcpyHostToDevice) );

  dim3 grid_dim(g, 1, 1);
  dim3 block_dim(h, 1, 1);

  kernel<<<grid_dim, block_dim>>>(n, dv_arr);



  checkCudaErrors( hipMemcpy(arr, dv_arr, bc_arr, hipMemcpyDeviceToHost) );

  for(int i = 0; i < n; i++)
    printf("[%d] = %f\n", i, arr[i]);

  checkCudaErrors( hipFree(dv_arr) );

  checkCudaErrors(hipDeviceReset());

}

*/
